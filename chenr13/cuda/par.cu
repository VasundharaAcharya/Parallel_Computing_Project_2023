
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

extern "C" {
    void computeMedoids(double* data, int* labels, double* medoids, int rank, int size);
    void findclosestmedoids(double *data, double *medoids, int *idx , int rank, int process_job,int size, int si,int ei);
}

#define K 10
#define n 7129
#define nf 34
#define REDUNDANT_SIZE 100


#ifndef MIN
#define MIN(x, y) ((x < y) ? x : y)
#endif


//here i am doing a reduction to find an array that has the minimum element and minimum element index for each block
__global__ void find_min(double *m_arr, int *mind, double *mval)
{
    // shared data for each block
    __shared__ double sdata[1024 + 10];
    __shared__ int sind[1024 + 10];
    int tid = threadIdx.x;
    size_t idx = threadIdx.x + blockDim.x * blockIdx.x;
    // putting data into shared memory
    if(idx<n)
    {
        sdata[tid]=m_arr[idx];
        if(m_arr[idx]==INFINITY)
            sind[tid]=-1;
        else
            sind[tid]=idx;
    }
    else
    {
        sdata[tid]=INFINITY;
        sind[tid]=-1;
    }

    __syncthreads();


    // reduction part 
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
    {
        __syncthreads();
        if (tid < s) // parallel sweep reduction
            if(sdata[tid]>=sdata[tid + s])
            {
                sdata[tid] = MIN(sdata[tid],sdata[tid + s]);
                sind[tid] = sind[tid+s];
            }
    }

    //if the thread id is 0, I am writing it to the min value and min index array
    if (tid == 0){mval[blockIdx.x]=sdata[0]; mind[blockIdx.x]=sind[0];}
}



//here i am making an array for average distances where each index i correcsponds to the average distance of point i
//I have these conditions while filling the array
//case 1: if two points have same cluster index, i am adding the distance to the point in consideration's avg distance
//case 2: if 2 points have different cluster index, i am giving the value of infinity
__global__ void find_avg_dist(double *data, int k, int *labels, double *avgs)
{
    int ind = blockIdx.x * blockDim.x + threadIdx.x;
    double as = 0;
    int c = 0;

    if (ind < n)
    {

        if (labels[ind] != k)
        {
            avgs[ind] = INFINITY;
        }

        else
        {

            for (int i = 0; i < n; i++)
            {
                if (i != ind and labels[i] == k)
                {
                    c += 1;
                    double ms = 0;

                    for (int j = 0; j < nf; j++)
                    {
                        ms += fabs(*(data + nf * ind + j) - *(data + nf * i + j));
                    }
                    as += ms;
                }
                else
                {
                    avgs[ind] = INFINITY;
                }
            }
            if (c == 0)
            {
                avgs[ind] = INFINITY;
            }
            else
            {
                avgs[ind] = as / c;
            }
        }
    }
    else
    {
        avgs[ind]=INFINITY;
    }
}

__global__ void findclosestmedoids_kernel(double *data, double *medoids, int *ids,int si,int ei)
{
    int i = si+( blockIdx.x*blockDim.x + threadIdx.x);
    if(i<ei+( blockIdx.x*blockDim.x + threadIdx.x))
    {
        double md=INFINITY;

        for(int j=0;j<K;j++)
        {
            double s=0;
            for(int k=0;k<nf;k++)
                s+=abs(*(data+i*nf+k)-*(medoids + j*nf+k));
            
            if(s<md)
            {
                md=s;
                *(ids+i)=j;
            }
        }
    }
}


void computeMedoids(double* data, int* labels, double* medoids, int rank, int size) 
{
    //here i am initializing the 
    int blockSize = 1024;
    int nblocks = (n+blockSize-1)/blockSize;
    int cudaDeviceCount;
    hipError_t cE;


    if( (cE = hipGetDeviceCount( &cudaDeviceCount)) != hipSuccess )
    {
    printf(" Unable to determine cuda device count, error is %d, count is %d\n",
    cE, cudaDeviceCount );
    exit(-1);
    }
    if( (cE = hipSetDevice( rank % cudaDeviceCount )) != hipSuccess )
    {
    printf(" Unable to have rank %d set to cuda device %d, error is %d \n",
    rank, (rank % cudaDeviceCount), cE);
    exit(-1);
    }

    // Divide the K clusters across the processes
    int medoids_per_proc = (K + size - 1) / size; // 2 = 15 / 6
    int remainder = K % size; // 4 = 10 % 6
    int si = rank * medoids_per_proc;
    int ei = si + medoids_per_proc;
    if (rank >= remainder) 
    {
        int diff = rank - remainder;
        si -= diff;
        ei = si + K / size;
    }
   

    #ifdef DEBUG_CUDA
    printf("  rank %d: computeMedoids(): cudaDeviceCount = %d,  medoids_per_proc = %d,  si = %d,  ei = %d\n", 
        rank, cudaDeviceCount, medoids_per_proc, si, ei);
    #endif

    double *avgs, *mval;
    int *mind;

    hipMallocManaged(&avgs, nblocks*blockSize*sizeof(double) + REDUNDANT_SIZE*sizeof(double));
    hipMallocManaged(&mind, nblocks*sizeof(int) + REDUNDANT_SIZE*sizeof(int));
    hipMallocManaged(&mval, nblocks*sizeof(double) + REDUNDANT_SIZE*sizeof(double));

        // double *avgs = (double*)malloc(nblocks*blockSize*sizeof(double));
        // int *mind = (int*)malloc(nblocks*sizeof(int));
        // double *mval =  (double*)malloc(nblocks*sizeof(double));

    for(int i=si;i<ei;i++)
    {
        // here for each cluster I am first computing the avg distance array and then I am finding the min index

        find_avg_dist<<<nblocks,blockSize>>>(data, i, labels,avgs);
    #ifdef DEBUG_CUDA
    printf("    rank %d: computeMedoids(): i = %d: finish find_avg_dist()\n", 
        rank, i);
    #endif
        hipDeviceSynchronize();



        find_min<<<nblocks,blockSize>>>(avgs, mind, mval);
    #ifdef DEBUG_CUDA
    printf("    rank %d: computeMedoids(): i = %d: finish find_min()\n", 
        rank, i);
    #endif
        hipDeviceSynchronize();
        


        // finding min index from the blocksize array 
        double mv= INFINITY;
        int mi;
        for(int j=0;j<nblocks;j++)
        {
            if(mval[j]<mv)
            {
                mv=mval[j];
                mi=mind[j];
            }
        }

        
    #ifdef DEBUG_CUDA
    printf("    rank %d: computeMedoids(): i = %d: mv = %.03lf, mi = %d\n", 
        rank, i, mv, mi);
    #endif

        // printf("index of medoid:%d cluster index:%d\n",mi,i);

        // printf("mval:\n");
        // for(int h=0;h<nblocks;h++)
        // {
        //     printf("%lf ",mval[h]);
        // }
        // printf("\n");


        // printf("mind:\n");
        // for(int h=0;h<nblocks;h++)
        // {
        //     printf("%d ",mind[h]);
        // }
        // printf("\n");

        //assigning the medoid for the particular cluster index
        if(mi!=-1)
        {
        for(int j=0;j<nf;j++)
            *(medoids + nf*i+j)=*(data + nf*mi+j);
        }

        

        
    }

        hipFree(avgs);
        hipFree(mind);
        hipFree(mval);

}

void findclosestmedoids(double *data, double *medoids, int *idx , int rank, int process_job,int size, int si,int ei)
{
    int blockSize = 1024;
    // printf("%d %d\n",si,ei);
    int size1 = (ei-si) + 1;
    int nblocks = (process_job+blockSize-1)/blockSize;
    int cE,cudaDeviceCount;

    if( (cE = hipGetDeviceCount( &cudaDeviceCount)) != hipSuccess )
    {
    printf(" Unable to determine cuda device count, error is %d, count is %d\n",
    cE, cudaDeviceCount );
    exit(-1);
    }
    if( (cE = hipSetDevice( rank % cudaDeviceCount )) != hipSuccess )
    {
    printf(" Unable to have rank %d set to cuda device %d, error is %d \n",
    rank, (rank % cudaDeviceCount), cE);
    exit(-1);
    }


    findclosestmedoids_kernel<<<nblocks,blockSize>>>(data, medoids, idx,si,ei);
    hipDeviceSynchronize();


    // printf("done\n");







}