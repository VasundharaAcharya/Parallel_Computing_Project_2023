
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define K 10
#define n 7129
#define nf 34


#ifndef MIN
#define MIN(x, y) ((x < y) ? x : y)
#endif


//here i am doing a reduction to find an array that has the minimum element and minimum element index for each block
__global__ void find_min(double *m_arr, int *mind, double *mval)
{
    // shared data for each block
    __shared__ double sdata[1024];
    __shared__ int sind[1024];
    int tid = threadIdx.x;
    size_t idx = threadIdx.x + blockDim.x * blockIdx.x;
    // putting data into shared memory
    if(idx<n)
    {
        sdata[tid]=m_arr[idx];
        sind[tid]=idx;
    }
    else
    {
        sdata[tid]=INFINITY;
        sind[tid]=-1;
    }

    __syncthreads();


    // reduction part 
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
    {
        __syncthreads();
        if (tid < s) // parallel sweep reduction
            if(sdata[tid]>=sdata[tid + s])
            {
                sdata[tid] = MIN(sdata[tid],sdata[tid + s]);
                sind[tid] = sind[tid+s];
            }
    }

    //if the thread id is 0, I am writing it to the min value and min index array
    if (tid == 0){mval[blockIdx.x]=sdata[0]; mind[blockIdx.x]=sind[0];}
}



//here i am making an array for average distances where each index i correcsponds to the average distance of point i
//I have these conditions while filling the array
//case 1: if two points have same cluster index, i am adding the distance to the point in consideration's avg distance
//case 2: if 2 points have different cluster index, i am giving the value of infinity
__global__ void find_avg_dist(double *data, int k, int *labels, double *avgs)
{
    int ind = blockIdx.x*blockDim.x + threadIdx.x;
    double as=0;
    int c = 0;

    if (labels[ind] != k)
    {
        avgs[ind] = INFINITY;
    }

    else
    {

        for (int i = 0; i < n; i++)
        {
            if (i != ind and labels[i] == k)
            {
                c += 1;
                double ms = 0;

                for (int j = 0; j < nf; j++)
                {
                    ms += fabs(*(data + nf * ind + j) - *(data + nf * i + j));
                }
                as += ms;
            }
            else
            {
                avgs[ind] = INFINITY;
            }
        }
        if (c == 0)
        {
            avgs[ind] = INFINITY;
        }
        else
        {
            avgs[ind] = as / c;
        }
    }
}



__global__ void findclosestmedoids_ker(double *data, double *medoids, int *med_labels,int si,int ei)
{
    int max_ind;
    int i = si+( blockIdx.x*blockDim.x + threadIdx.x);

    if(i<=ei)
    {
        double md=INFINITY;

        for(int j=0;j<K;j++)
        {
            double s=0;
            for(int k=0;k<nf;k++)
                s+=abs(*(data+i*nf+k)-*(medoids + j*nf+k));
            
            if(s<md)
            {
                md=s;
                max_ind = j;
            }
        }

        med_labels[i]=max_ind;
    }
}


extern "C" void computeMedoids(double* data, int* labels, double* medoids, int rank, int size) 
{
    int blockSize = 1024;
    int nblocks = (n+blockSize-1)/blockSize;
    int cE,cudaDeviceCount;

    if( (cE = hipGetDeviceCount( &cudaDeviceCount)) != hipSuccess )
    {
    printf(" Unable to determine cuda device count, error is %d, count is %d\n",
    cE, cudaDeviceCount );
    exit(-1);
    }
    if( (cE = hipSetDevice( rank % cudaDeviceCount )) != hipSuccess )
    {
    printf(" Unable to have rank %d set to cuda device %d, error is %d \n",
    rank, (rank % cudaDeviceCount), cE);
    exit(-1);
    }

    // Divide the K clusters across the processes
    int medoids_per_proc = K / size;
    int remainder = K % size;
    int si = rank * medoids_per_proc;
    int ei = si + medoids_per_proc;
   
    if (rank == size - 1) {
        ei += remainder;
    }

    double *avgs = (double*)malloc(n*sizeof(double));
    int *mind = (int*)malloc(nblocks*sizeof(int));
    double *mval =  (double*)malloc(nblocks*sizeof(double));


    for(int i=si;i<ei;i++)
    {
        // here for each cluster I am first computing the avg distance array and then I am finding the min index
        find_avg_dist<<<nblocks,blockSize>>>(data, i, labels,avgs);
        hipDeviceSynchronize();
        find_min<<<nblocks,blockSize>>>(avgs, mind,mval);
        hipDeviceSynchronize();

        // finding min index from the blocksize array 
        double mv= INFINITY;
        int mi;
        for(int j=0;j<nblocks;j++)
        {
            if(mval[j]<mv)
            {
                mv=mval[j];
                mi=mind[j];
            }
        }

        //assigning the medoid for the particular cluster index
        for(int j=0;j<nf;j++)
            *(medoids + nf*i+j)=*(data + nf*mi+j);
        
    }


        printf("done\n");






}

extern "C"  void findclosestmedoids(double *data, double *medoids, int *idx, int rank, int size,int process_job, int si,int ei)
{
    int blockSize = 1024;
    int size1 = (ei-si) + 1;
    int nblocks = (size1+blockSize-1)/blockSize;
    int cE,cudaDeviceCount;

    printf("%d %d\n",si,ei);
    if( (cE = hipGetDeviceCount( &cudaDeviceCount)) != hipSuccess )
    {
    printf(" Unable to determine cuda device count, error is %d, count is %d\n",
    cE, cudaDeviceCount );
    exit(-1);
    }
    if( (cE = hipSetDevice( rank % cudaDeviceCount )) != hipSuccess )
    {
    printf(" Unable to have rank %d set to cuda device %d, error is %d \n",
    rank, (rank % cudaDeviceCount), cE);
    exit(-1);
    }

    findclosestmedoids_ker<<<nblocks,blockSize>>>(data, medoids, idx,si,ei);

    hipDeviceSynchronize();





}


