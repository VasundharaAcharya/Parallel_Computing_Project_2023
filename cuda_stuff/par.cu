#include "hip/hip_runtime.h"

/* This code uses KMedoids implemented with Manhattan distance metric to compute the cluster assignments.
As per the requirement of the project, the training file is converted to bin format and is read and written using MPI I/O.
*/



#include<stdio.h>
#include<stdlib.h>
#include<unistd.h>
#include<stdbool.h>
#include<time.h>
#include<mpi.h>
#include<string.h>
#include<math.h>
#include "clockcycle.h"
#define Genes 15  // X Total Number of genes to be given as an input. 
#define Samples 34     // Represents the sample genes
//Change the value of K to obtain the results with different clusters
#define K 4 // Number of clusters
//Initializations
int *cluster_idx;             
double *gene_data;          
double *medoids;   					 // pointer to data  which stores the index of the centroid nearest to each pixel


//Define a macro to compute the minimum
#ifndef MIN1
#define MIN1(x, y) ((x < y) ? 1: 0)
#endif


//Macro
#ifndef MIN
#define MIN(x, y) ((x < y) ? x : y)
#endif

//Macro to compute the maximum
#ifndef MAX
#define MAX(x, y) ((x < y) ? y : x)
#endif


/*INFINITY is a macro constant defined in the <math.h> library, and we use it to perform mathematical comparisons*/
#ifdef INFINITY
/* INFINITY is supported */
#endif

// void computeMedoids(double* data, int* labels, double* medoids, int rank, int size);
// void findclosestmedoids(double *data, double *medoids, int *idx, int rank, int size,int process_job, int si,int ei);
void computeMedoids(double* data, int* labels, double* medoids, int rank, int size);
void findclosestmedoids(double *data, double *medoids, int * , int rank, int process_job,int size, int si,int ei);
//Finding the closeset medoids
//This function works totally fine
// void findclosestmedoids(double *num, double *medoids, int *idx, int rank, int process_job,int size,int si,int ei) {
//     int i, j, l, for_i;
//     double sum, dist[K], min_dist, local_min_dist;
//     // Broadcast the medoids to all processes
//     MPI_Bcast(medoids, K * Samples, MPI_DOUBLE, 0, MPI_COMM_WORLD);

//     // Find the closest medoid for each local data point
//     for (for_i = 0; for_i <process_job; for_i++) {
//         i = si + for_i;
//         local_min_dist = INFINITY;

//         for (j = 0; j < K; j++) {
//             sum = 0;

//             for (l = 0; l < Samples; l++) {
             
//                 sum += fabs(num[i * Samples + l] - medoids[j * Samples + l]);
//             }

//             dist[j] = sum;

//             if (MIN1(dist[j],local_min_dist)) {
//                 local_min_dist = dist[j];
//                 idx[i] = j;
//             }
//         }


//     }



    // Reduce the local min distances to find the global min distance
//     //The reason why we use
//     MPI_Allreduce(&local_min_dist, &min_dist, 1, MPI_DOUBLE, MPI_MIN, MPI_COMM_WORLD);

//     // Update the indices for data points with the global min distance
//     for (i = si; i < ei; i++) {
//         if (dist[idx[i]] == min_dist) {
//             idx[i] = j;
//         }
//     }
// }





// //This code is same as the serial code, but the difference is the splitting of the clusters among processes
// void computeMedoids(double* gene_data, int* idx, double* medoids, int rank, int size) {
//     int i, j, k, l, m;
//     double min_distance, distance, sum, temp;

//     // using entire gene data instead of sampling
//     int sample_size = Genes; 
   
//     // Divide the K clusters across the processes
//     int medoids_per_proc = K / size;
//     int remainder = K % size;
//     int start_idx = rank * medoids_per_proc;
//     int end_idx = start_idx + medoids_per_proc;
   
//     if (rank == size - 1) {
//         end_idx += remainder;
//     }

//     //printf("Process %d: start_idx = %d, end_idx = %d\n", rank, start_idx, end_idx);

//     // Seed the random number generator with the rank
//     srand(time(0) + rank);

//     for (i = start_idx; i < end_idx; i++) {
//         min_distance = 1e9;

//         // Calculate distances for all points in the cluster
//         for (j = 0; j < Genes; j++) {
//             if (idx[j] == i) {
//                 sum = 0.0;
//                 for (k = 0; k < Genes; k++) {
//                     if (idx[k] == i) {
//                         distance = 0.0;
//                         for (l = 0; l < Samples; l++) {
//                             // manhattan distance
//                             double diff = *(gene_data + j * Samples + l) - *(gene_data + k * Samples + l);
//                             distance += fabs(diff);
//                         }
//                         sum += distance;
//                     }
//                 }
// //I am using the macro i defined for this
//                 if (MIN1(sum, min_distance)) {
//                     min_distance = sum;
//                     for (m = 0; m < Samples; m++) {
//                         temp = *(gene_data + j * Samples + m);
//                         *(medoids + i * Samples + m) = temp;
//                     }                    
//                 }
//             }
//         }
//     }
// }








int main(int argc, char *argv[]){

          int myrank, numranks, result;
          int i,j,k;
          int process_job, each_chunk_pos;
          double starttime, endtime;

          MPI_Init(NULL, NULL);
          MPI_Status status;
          MPI_Request r_request, s_request;

          int world_size;
          MPI_Comm_size(MPI_COMM_WORLD, &world_size);

          int world_rank;
          MPI_Comm_rank(MPI_COMM_WORLD, &world_rank);

          int rnd_num;
          int n =Genes, fs = Samples, k1=K, *label,lab;
          MPI_Request request1, request2, request3, request4; 
          double num1;
          MPI_File fh,fh1;
          FILE *fp;
          int si,ei;

          //Determination of each chunk of the input file each process is going to read
          process_job=Genes/world_size;
          if(world_rank==world_size-1) {
              process_job=process_job+Genes%world_size;
          }

          gene_data = (double*) calloc(Genes * Samples, sizeof(double));
          medoids = (double*) calloc(K * Samples, sizeof(double));
          cluster_idx = (int*) calloc(Genes, sizeof(int));

  







/* MPI I/O to read from the bin file created*/
          int resultant;
          //For this we need to have the bin file rather than txt file.

          resultant = MPI_File_open(MPI_COMM_WORLD, "input.bin", MPI_MODE_RDONLY, MPI_INFO_NULL, &fh);
          if (resultant != MPI_SUCCESS) {
              printf("Error opening file. \n");
              exit(-1);
          }




          // Determining the file size is the most important step as I experimented with partial filesize led to wrong results
          MPI_Offset file_size;
          MPI_File_get_size(fh, &file_size);

         //This is taken from slide number 11 of mpi I/O
          MPI_Offset buffsize = file_size / world_size;
          MPI_Offset start_idx = world_rank * buffsize;
          MPI_Offset end_idx =world_rank * buffsize + buffsize;
          if (world_rank == world_size - 1) {
              end_idx = file_size;

          }
          MPI_Offset local_size =world_rank * buffsize + buffsize -world_rank * buffsize ;

          double* local_data = (double*)malloc(local_size * sizeof(double));

          // This is taken from the slide number 20 of the mpi I/O slide
         // MPI_File_set_view(fh,  world_rank * buffsize, MPI_DOUBLE, MPI_DOUBLE, "native", MPI_INFO_NULL);


          MPI_File_read(fh, local_data, local_size, MPI_DOUBLE, &status);

          MPI_File_close(&fh);






for (i = 0; i < Genes; i++) {
    for (j = 0; j < Samples; j++) {
        *(gene_data + i * Samples + j) = *(local_data + i * Samples + j);
    }
}



free(local_data);

  	if (world_rank==0)
	{	
    
           
              int lower =0;
              int upper =process_job-1;
              srand(time(0));

  
  // Start the timer here
    starttime = clock_now();
//Random initialize of medoids takes place here
		for (int i = 0; i < K; i++) {

			int rnd_num = (rand()%(upper-lower + 1)) + lower;
	
		
			for (j=0;j<Samples;j++){ 
        		*(medoids+i*Samples+j) = *(gene_data+rnd_num*Samples+j); 
        	} 
      
    }
  
  }
MPI_Barrier(MPI_COMM_WORLD);

   
 /*This is for the findclosestmedoids function. We have computed the starting and ending indexes separately 
 for the computeMedoids function inside the function.*/

    si = (world_rank)*(Genes/world_size);

    if(world_rank==(world_size-1))
    {
        ei = Genes;
    }
    else
    {
        ei = (world_rank+1)*(Genes/world_size);
    }

    
 /*We run the findclosestmedoids for 10 iterations. 
 The broadcast to all the processes and also the computation of final result using MPI_Allreduce is done in
 the function itself. Therefore we do not do it here. But we meed to do all gather to gather the results from all processes after
 the computeMed function */



	//MPI_Bcast(medoids, K*Samples, MPI_DOUBLE, 0, MPI_COMM_WORLD);
 
      for (i=0;i<1;i++){
        printf("process_job:%d\n",process_job);

        findclosestmedoids((double *)gene_data, (double *)medoids, &cluster_idx[0],world_rank,process_job,world_size,si,ei);
        MPI_Barrier(MPI_COMM_WORLD);
        computeMedoids((double *)gene_data, &cluster_idx[0], (double *)medoids,world_rank, world_size);
        MPI_Barrier(MPI_COMM_WORLD);


        /*All gather must be done here. The reason for using MPI_IN_PLACE is because the send 
      and recieve buffer are the same */

	MPI_Allgather(MPI_IN_PLACE, 0, MPI_DATATYPE_NULL, medoids, K * Samples /world_size, MPI_DOUBLE, MPI_COMM_WORLD);
//Since I am using all gather and updating the medoids, I dont require the below updation logic.

	//MPI_Barrier(MPI_COMM_WORLD);

   /* for (int a=0; a<K;a++){
        
          for (int b=0;b<Samples;b++){
            *(medoids+a*Samples+b)=*(medoids+a*Samples+b)/world_size;
          }
        }
      */
	endtime = clock_now();


      }
      
    // Print the medoids computed by the root process
    if (world_rank == 0) {

        

        printf("Total time taken to run K-Medoids on %d genes with %d clusters is %e seconds.\n", Genes, K, (endtime-starttime)/512000000.0f);
        printf("Total clockcycles taken to run K-Medoids on %d genes with %d clusters  is %f cycles.\n", Genes, K, (endtime-starttime));
        printf("Medoids:\n");
        for (i = 0; i < K; i++) {
            printf("Medoid %d: ", i + 1);
            for (j = 0; j < Samples; j++) {
                printf("%lf ", *(medoids + i * Samples + j));
            }
            printf("\n");
        }


    // printf("Cluster assignments:\n");
    // for (i = 0; i < Genes; i++) {
    //     printf("Data point %d assigned to cluster %d\n", i, cluster_idx[i]);
    // }




 /* MPI fileWRITE takes place here */
    // Write clustered gene data to file
        MPI_File clustered_gene_file;
        MPI_File_open(MPI_COMM_SELF, "Gene_clusters.bin", MPI_MODE_CREATE | MPI_MODE_WRONLY, MPI_INFO_NULL, &clustered_gene_file);
        for (i = 0; i < K; i++) {
            for (j = 0; j < Samples; j++) {
                MPI_File_write(clustered_gene_file, &gene_data[i*Samples+j], 1, MPI_DOUBLE, MPI_STATUS_IGNORE);
            }
        }
        // Close the output file
        MPI_File_close(&clustered_gene_file);


        MPI_File output_file;
        MPI_File_open(MPI_COMM_SELF, "output_gene_medoids__manhattan_clusters.bin", MPI_MODE_CREATE | MPI_MODE_WRONLY, MPI_INFO_NULL, &output_file);
        for (i = 0; i < K; i++) {
            for (j = 0; j < Samples; j++) {
                MPI_File_write(output_file, &medoids[i * Samples + j], 1, MPI_DOUBLE, MPI_STATUS_IGNORE);
            }
        }

        // Close the output file
        MPI_File_close(&output_file);



       //Write the cluster assignments to the file
        MPI_File cluster_idxs;
        MPI_File_open(MPI_COMM_SELF, "cluster_assignment.bin", MPI_MODE_CREATE | MPI_MODE_WRONLY, MPI_INFO_NULL, &cluster_idxs);
        for (i = 0; i < Genes; i++) {
            MPI_File_write(cluster_idxs, &cluster_idx[i], 1, MPI_INT, MPI_STATUS_IGNORE);
        }
        MPI_File_close(&cluster_idxs);



       
        }







        // Free allocated memory
        if (world_rank == 0) {
            free(gene_data);
            free(cluster_idx);
            free(medoids);
        }

        MPI_Finalize();
        

      
        return 0;
    }
